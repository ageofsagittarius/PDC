

#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;


__global__ void func(double *one,int * two,int *three,double *four,double *five,int row)
{   
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int jump = blockDim.x * gridDim.x;
    
    for (int i = start; i < row; i += jump) {
        double sum = 0.0;
    
        for (int j = two[i]; j < two[i+1]; j++) {
            sum += one[j] * five[three[j]];
        }
        
        four[i] = sum;
    }
}

int main()
{
    ifstream matrixStream;
    matrixStream.open("../input/assignment9-dataset/inputfile.mtx");

    // ignoring the unnecessary things
    while (matrixStream.peek() == '%')
        matrixStream.ignore(2048, '\n');

    int row, column, count_v, r, c;
    double k;

    // taking row col and number of nonzero elements
    matrixStream >> row >> column >> count_v;

    cout << "Rows: " << row << " "
         << "Columns: " << column << " Non zero elements: " << count_v << endl;

    double **matrix = (double **)malloc(row * sizeof(double *));

    for (int i = 0; i < row; i++)
    {
        matrix[i] = (double *)malloc(column * sizeof(double));
    }

    // reading the elemnts of the matrix
    for (int i = 0; i < row; i++)
    {
        matrixStream >> r >> c >> k;
        r--;
        c--;
        matrix[r][c] = k;
    }

    // reading from the elements
    // reading from txt file
    ifstream vector_stream;
    vector_stream.open("../input/assignment9-dataset/vector.txt");

    double *in_vector = (double *)malloc(row * sizeof(double));

    string data = "";
    for (int i = 0; i < row; i++)
    {
        getline(vector_stream, data, ',');
        in_vector[i] = stod(data);
    }
    
    printf("The given vector is: \n");
    for(int i = 0; i < column; i++){
        cout<<in_vector[i]<<", ";
    }
    cout<<endl<<endl;
    

    double *matrixValues = (double *)malloc(count_v * sizeof(double));
    int *rMatrix = (int *)malloc(row + 1 * sizeof(int));
    int *cMatrix = (int *)malloc(count_v * sizeof(int));
    double *answer = (double *)malloc(row * sizeof(double));
    
    for (int i = 0; i < count_v; i++)
    {
        matrixValues[i] = 0;
        cMatrix[i] = 0;
    }
    int ind = 0;
    int cnt = 0;
    rMatrix[0] = 0;

    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < column; j++)
        {
            if (matrix[i][j] != 0)
            {
                matrixValues[ind++] = matrix[i][j];
                cnt++;
            }
        }

        rMatrix[i + 1] = cnt;
    }

    for (int i = 0; i < count_v; i++) cMatrix[i] = i;
    
    double *one = (double *)malloc(count_v * sizeof(double));
    int *two = (int *)malloc(row + 1 * sizeof(int));
    int *three = (int *)malloc(count_v * sizeof(int));
    double *four = (double *)malloc(row * sizeof(double));
    double *five = (double *)malloc(row * sizeof(double));
    
    // Allocate device memory 
    hipMalloc((void**)&one, sizeof(double) * count_v);
    hipMalloc((void**)&two, sizeof(int) * row+1);
    hipMalloc((void**)&three, sizeof(int) * count_v);
    hipMalloc((void**)&four, sizeof(double) * row);
    hipMalloc((void**)&five,sizeof(double)*row);
    
    // Transfer data from host to device memory
    hipMemcpy(one, matrixValues, sizeof(double) * count_v, hipMemcpyHostToDevice);
    hipMemcpy(two, rMatrix, sizeof(int) * row+1, hipMemcpyHostToDevice);
    hipMemcpy(three, cMatrix, sizeof(int) * count_v, hipMemcpyHostToDevice);
    hipMemcpy(four, answer, sizeof(double) * row, hipMemcpyHostToDevice);
    hipMemcpy(five, in_vector, sizeof(double) * row, hipMemcpyHostToDevice);
    
    // Executing kernel 
    int size_k_b = row;
    int size_g_k = 1;
    func<<<size_g_k,size_k_b>>>(one, two,three,four,five,row);
    
    //Transfer data back to host memory
    hipMemcpy(answer, four, sizeof(double) * row, hipMemcpyDeviceToHost);
    
    printf("The CSR format : \n");
    printf("The element array is: \n");
    for(int i = 0; i < count_v; i++){
        cout<<matrixValues[i]<<", ";
    }
    cout<<endl<<endl;
    
    printf("The column array is: \n");
    for(int i = 0; i < count_v; i++){
        cout<<cMatrix[i]<<", ";
    }
    cout<<endl<<endl;
    
    printf("The non-zero array is: \n");
    for(int i = 0; i < count_v; i++){
        cout<<rMatrix[i]<<", ";
    }
    cout<<endl<<endl;
    
    double seq[row];
    
    for (int i = 0; i < row; i++) {
        double sum = 0.0;
        for (int j = rMatrix[i]; j < rMatrix[i+1]; j++) {
            sum += matrixValues[j] * in_vector[cMatrix[j]];
        }
        seq[i] = sum;
    }
    
    printf("The resultant vector found using sequential is: \n");
    for(int i = 0; i < row; i++){
        cout<<seq[i]<<", ";
    }
    cout<<endl<<endl;
    
    printf("The resultant vector found using parallel(cuda) is: \n");
    for(int i = 0; i < row; i++){
        cout<<answer[i]<<", ";
    }
    cout<<endl<<endl;
    
    
    bool flag = true;
    
    for(int i = 0; i < row; i++){
        if(seq[i] != answer[i]){
            flag = false;
            break;
        }
    }
    
    if(flag) cout<<"CORRECT the output of CUDA matches with that of sequential";
    else cout<<"INCORRECT the output doesn't match";
    
    
    return 0;
}
