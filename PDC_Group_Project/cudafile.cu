
/*
    PDC Group Project: 2022
    Ritik Mehra
    Abhishek Bhaware
    Rajat Mehra
    Sarvesh
*/


#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

// kernel function
__global__ void fun1(double *matrix, int N, int idx){

    for(int i = idx * N + idx + 1 ; i < idx * N + N; i++){
        matrix[i] = (matrix[i] / matrix[idx * N + idx]);
    }
}

//kernel function that uses shared memory for the LU Calculation and storing result in row-array format
__global__ void fun2(double *mat, int N, int idx, int size){
    extern __shared__ double sharedMem[];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    if(tid == 0){
        for(int i = idx; i < N; i++){
            sharedMem[i] = mat[(idx * N) + i];
        }
    }
    
    __syncthreads();
    
    int arrayId = (size * bid) + tid; 
    
    int curIdx = arrayId * N;
    int left = curIdx + idx, right = curIdx + N;
    
    if(curIdx > idx * N){
        for(int i = left + 1; i < right; i++){
            mat[i] -= mat[left] * sharedMem[i - curIdx];
        }
    }
    
    __syncthreads();
    
    }

//function to print the matrices
void printMat(double** mat, int N){
    int i, j;
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%.3f\t", mat[i][j]);
        }
        cout<<endl;
    }
}

//function used for matrix multiplication
void mulMat(double** mat1, double**mat2, double** res, int N){
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            res[i][j] = 0;
            for (int k = 0; k < N; k++) {
                res[i][j] += mat1[i][k] * mat2[k][j];
            }
        }
    }
}

int main(){
    //the size of the matrix
    int N = 10;
    cout<<"The dimension of the matrix is: "<<N<<endl;
    //allocating memory on host
    double *input = new double[N * N];
    double *output = new double[N * N];
    
    for (int i = 0; i <= (N * N); i++){
        input[i] =(rand() % 10) + 1;
    }
    
    int cnt = 0;
    double sum = 0;
    for(int i = 0; i < N; i++, cnt++){
        for(int j = i*N; j < i*N + N; j++){
            sum += abs(input[j]);
        }
        sum -= abs(input[i*N + cnt]);
        input[i*N + cnt] = sum + (rand() % 5) + 1;
        sum = 0;
    }
    
    //starting time
    clock_t begin = clock();
    
    // allocate memory on device i.e. GPU
    double *mat;
    hipMalloc(&mat, N * N * sizeof(double));
    
    // copy from host to device
    hipMemcpy(mat, input, N * N * sizeof(double), hipMemcpyHostToDevice);
    
    int numOfThreads = 8;
    int numOfBlock = N / numOfThreads;
    if(N % numOfThreads){
        numOfBlock++;
    }

    for(int i = 0; i < N; i++){
        fun1<<<1, 1>>>(mat, N, i);
        fun2<<<numOfBlock, numOfThreads, N * sizeof(double)>>>(mat, N, i, numOfThreads);
    }
    
    // copying back from device to host
    hipMemcpy(output, mat, N * N * sizeof(double), hipMemcpyDeviceToHost);
    
    //ending time
    clock_t end = clock();
    
    // declare and initialise the matrices
    double **matrix = new double*[N];
    double **mm = new double*[N];
    double **L = new double*[N];
    double **U = new double*[N];
    
    for(int i = 0; i < N; i++){
        matrix[i] = new double[N];
        mm[i] = new double[N];
        L[i] = new double[N];
        U[i] = new double[N];
    }
    
    //putting the values in the matrices from the output obtained
    
    //original matrix
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            mm[i][j] = input[i * N + j];
            matrix[i][j] = output[i * N + j];
        }
    }
    
    //filling the lower and upper diagonal matrix i.e L and U
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            for(int k = 0; k < N; k++){
                if(i >= k){
                    L[i][k] = output[i * N + k];
                }
                else{
                    L[i][k] = 0;
                }

                if(k == j){
                    U[k][j] = 1;
                }
                else if(k < j){
                    U[k][j] = output[k * N + j];
                }
                else{
                    U[k][j] = 0;
                }
            }
        }
    }
    
    cout<<"The original matrix is: \n";
    printMat(mm, N);
    
    //print the resultant matrices i.e. L and U
    cout<<"\nThe resultant matrices are:\n";
    cout<<"\nL Matrix is:\n";
    printMat(L, N);
    cout<<"\nU Matrix is:\n";
    printMat(U, N);
    
    // time taken to execute the program
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    
    printf("\nTime taken to evaluate using sequential  :%.6f Seconds\n", time_spent);
    
    // for the correctness of algorithm, we may multiply the matrix L and U and verify it with input matrices
    double **m1 = new double* [N];
    for(int i = 0; i < N; i++){
        m1[i] = new double[N];
    }
    mulMat(L, U, m1, N);
    cout<<"\n---------------------------\n";
    //cout << "Multiplication of given two matrices is:\n";
    //printMat(m1, N);


    int flag = 1;
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            if(abs(mm[i][j] - m1[i][j]) >= 0.05){
                flag = 0;
                break;
            }
        }
    }
    
    if(flag)cout<<"\nCORRECT\n";
    else cout<<"\nINCORRECT\n";

    
    hipFree(mat);
    
    
    return 0;
}
