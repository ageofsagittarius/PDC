
/*
    IIT2019038
    Ritik Mehra
    
    PDC Assignment 11: Implementation of histogramming parallel algorithms
*/


#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

int N = 1024;
int numOfBins = 16;
int segmentSize = (256 / numOfBins);


// parallel algorithm1
__global__ void algo1(int* input , int *output, int N, int segmentSize){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = input[id];
    atomicAdd(&output[idx / segmentSize] , 1);
}

void parallelAlgo1(int *input, int *answer){
    int *output = (int*) malloc(numOfBins * sizeof(int));
    
    for(int i = 0; i < numOfBins; i++){
        output[i] = 0;
    }
    
    clock_t begin = clock();
    printf("\n*** Algo1 ***\n");
    
    //Allocating the memory on the device
    int *arr, *ans;
    
    hipMalloc(&arr, sizeof(int) * N);
    hipMalloc(&ans, sizeof(int) * numOfBins);
    
    //Copying the array to the device
    hipMemcpy(arr, input, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(ans, output, sizeof(int) * numOfBins, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    int numOfThreads = 512;
    int blockSize = 2 * numOfThreads;
    int n;
    if(N % blockSize == 0){
        n = N;
    }
    else{
        n = (1 + N / blockSize) * blockSize;
    }
    int gridSize = n/blockSize;
    numOfThreads = N;
    algo1<<<gridSize, numOfThreads>>>(arr, ans, N, segmentSize);
    hipDeviceSynchronize();

    // Copy the result back
    hipMemcpy(output, ans, numOfBins * sizeof(int), hipMemcpyDeviceToHost);
    
    // ending time
    clock_t end = clock();
    
    printf("The resultant histogram using algo1 is: \n");
    for(int i = 0; i < numOfBins; i++){
        cout<<output[i]<<" \t";
    }
    cout<<endl;

    bool flag = true;
    for(int i = 0; i < numOfBins; i++){
        if(output[i] != answer[i]){
            flag = false;
            break;
        }
    }
    
    if(flag) printf("CORRECT\n");
    else printf("INCORRECT\n");
    
    //Free the allocated memory
    hipFree(arr);
    hipFree(ans);
    
    free(output);
    free(input);
    
    // Time taken for complete execution
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Execution Time  :%.6f Seconds\n", time_spent);
}


// parallel algorithm 2: Using global memory
// Both the input data and output array are stored in the GPU global memory area.
__global__ void algo2(int* input, int* output, int N, int segmentSize) {
    int blockId = blockIdx.x * blockDim.x;
    int threadId = threadIdx.x;
    
    // find the id
    int arrayId = blockId + threadId;
    
    int jump = gridDim.x * blockDim.x;
    
    int currIdx;
    for(int i = arrayId; i < N; i += jump) {
        // find the index
        currIdx = input[i];
        atomicAdd(&output[currIdx / segmentSize], 1);
    }
}

void parallelAlgo2(int *input, int *answer){
    int *output = (int*) malloc(numOfBins * sizeof(int));
    
    for(int i = 0; i < numOfBins; i++){
        output[i] = 0;
    }
    
    clock_t begin = clock();
    printf("\n*** Algo2 ***\n");
    
    //Allocating the memory on the device
    int *arr, *ans;
    
    hipMalloc(&arr, sizeof(int) * N);
    hipMalloc(&ans, sizeof(int) * numOfBins);
    
    //Copying the array to the device
    hipMemcpy(arr, input, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(ans, output, sizeof(int) * numOfBins, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    int numOfThreads = 512;
    int blockSize = 2 * numOfThreads;
    int n;
    if(N % blockSize == 0){
        n = N;
    }
    else{
        n = (1 + N / blockSize) * blockSize;
    }
    int gridSize = n/blockSize;
    
    algo2<<<gridSize, numOfThreads>>>(arr, ans, N, segmentSize);
    hipDeviceSynchronize();

    // Copy the result back
    hipMemcpy(output, ans, numOfBins * sizeof(int), hipMemcpyDeviceToHost);
    
    // ending time
    clock_t end = clock();
    
    printf("The resultant histogram using algo2 is: \n");
    for(int i = 0; i < numOfBins; i++){
        cout<<output[i]<<" \t";
    }
    cout<<endl;

    bool flag = true;
    for(int i = 0; i < numOfBins; i++){
        if(output[i] != answer[i]){
            flag = false;
            break;
        }
    }
    
    if(flag) printf("CORRECT\n");
    else printf("INCORRECT\n");
    
    //Free the allocated memory
    hipFree(arr);
    hipFree(ans);
    
    free(output);
    free(input);
    
    // Time taken for complete execution
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Execution Time  :%.6f Seconds\n", time_spent);
}


// parallel algorithm 3: Using shared memory
// Each thread here has its own private copy of the bucket array
__global__ void algo3(int* input, int* output, int *res, int N, int segmentSize, int numOfBins) {
    int blockId = blockIdx.x * blockDim.x;
    int threadId = threadIdx.x;
    
    // find the id
    int arrayId = blockId + threadId;
    
    // Initalize the shared memory to 0
    if (threadId < numOfBins) {
        res[threadId] = 0;
    }
    
    __syncthreads();
    
    int jump = gridDim.x * blockDim.x;
    
    for (int i = arrayId; i < N; i += jump) {
        //find the idx for histogram
        int currIdx = input[i];
        atomicAdd(&res[(currIdx / segmentSize)], 1);
    }
    
    __syncthreads();

    //store the results back
    if (threadId < numOfBins) {
        atomicAdd(&output[threadId], res[threadId]);
    }
}

void parallelAlgo3(int *input, int *answer){
    int *output = (int*) malloc(numOfBins * sizeof(int));
    
    for(int i = 0; i < numOfBins; i++){
        output[i] = 0;
    }
    
    printf("\n*** Algo3 ***\n");
    
    //Allocating the memory on the device
    int *arr, *ans, *res;
    
    hipMalloc(&arr, sizeof(int) * N);
    hipMalloc(&ans, sizeof(int) * numOfBins);
    hipMalloc(&res, sizeof(int) * numOfBins);
    
    //Copying the array to the device
    hipMemcpy(arr, input, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(ans, output, sizeof(int) * numOfBins, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    int numOfThreads = 512;
    int blockSize = 2 * numOfThreads;
    int n;
    if(N % blockSize == 0){
        n = N;
    }
    else{
        n = (1 + N / blockSize) * blockSize;
    }
    int gridSize = n/blockSize;
    
    clock_t begin = clock();
    
    algo3<<<gridSize, numOfThreads>>>(arr, ans, res, N, segmentSize, numOfBins);
    hipDeviceSynchronize();

    // Copy the result back
    hipMemcpy(output, ans, numOfBins * sizeof(int), hipMemcpyDeviceToHost);
    
    // ending time
    clock_t end = clock();
    
    printf("The resultant histogram for algo3 is: \n");
    for(int i = 0; i < numOfBins; i++){
        cout<<output[i]<<" \t";
    }
    cout<<endl;
    
    bool flag = true;
    for(int i = 0; i < numOfBins; i++){
        if(output[i] != answer[i]){
            flag = false;
            break;
        }
    }
    
    if(flag) printf("CORRECT\n");
    else printf("INCORRECT\n");
    
    //Free the allocated memory
    hipFree(arr);
    hipFree(ans);
    hipFree(res);
    
    free(output);
    free(input);
    
    // Time taken for complete execution
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Execution Time  :%.6f Seconds\n", time_spent);
}

int main() {
    
    // starting time
    clock_t begin = clock();
    
    printf("The size of the array is: %d \n", N);
    printf("#bins: %d \n", numOfBins);

    //Create pointers and allocate arrays
    int *input = (int*) malloc(N * sizeof(int));
    int *input1 = (int*) malloc(N * sizeof(int));
    int *input2 = (int*) malloc(N * sizeof(int));
    int *input3 = (int*) malloc(N * sizeof(int));
    
    //Fill the input array with numbers
    for(int i = 0; i < N; i++){ 
        input[i] = rand() % 256;
        input1[i] = input[i];
        input2[i] = input[i];
        input3[i] = input[i];
    }
    for(int i = 0; i < N; i++){
        input1[i] = input[i];
    }
    for(int i = 0; i < N; i++){
        input2[i] = input[i];
    }
    for(int i = 0; i < N; i++){
        input3[i] = input[i];
    }
    
    printf("The input array is: \n");
    for(int i = 0; i < N; i++){
        cout<<input[i]<<" \t";
    }
    cout<<endl;
    
    // sequential
    printf("\n*** Sequential ***\n");
    
    int *seqOutput = (int*) malloc(numOfBins * sizeof(int));
    
    for(int i = 0; i < N; i++){
        int currIdx = input[i] / segmentSize;
        seqOutput[currIdx] += 1;
    }
    
    printf("The resultant histogram for using sequential approach is: \n");
    for(int i = 0; i < numOfBins; i++){
        cout<<seqOutput[i]<<" \t";
    }
    cout<<endl;
    
    // ending time
    clock_t end = clock();
    // Time taken for complete execution
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Execution Time  :%.6f Seconds\n", time_spent);
    
    
    // parallel algorithm 1
    parallelAlgo1(input1, seqOutput);
    
    
    // parallel algorithm 2
    parallelAlgo2(input2, seqOutput);
    
    
    // parallel algorithm 2
    parallelAlgo3(input3, seqOutput);
    
    
    return 0;
}

/*
    The output for all the three algorithms does match with that of the sequential and hence is "CORRECT".
    
    The fastest algorithm is the Parallel Algo 3. It took 0.000030 Seconds to execute as compared to algo2 
    and algo1 which took 0.000204 Seconds and 0.172672 Seconds to execute.
    
    In the algorithm 3 it uses, the GPU shared memory. Every thread is given its own copy of the result
    array. The input array is stored in the global memory and the thread's own result array is stored in the
    shared memory that leads to faster calculation. Finally when all the threads are done with processing and are synchronized the final
    result is updated with the sum of all of the results obtained from the threads.
*/
