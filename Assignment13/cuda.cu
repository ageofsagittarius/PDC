/*
    IIT2019038
    Ritik Mehra
    PDC Assignment-13

    Device Specifications -
		Intel(R) Core(TM) i5-8265U CPU @ 1.60GHz
		RAM               :  8 GB
		CPU               :  8
		Threads per core  :  2
    
    Problem Statement: Find the value of Pi using the Dart Board Method
    
    Instructions to run:
        mpic++ mpi-c .c -o mpi.o
        nvcc -c cuda.cu -o cuda.o
        mpic++ mpi.o cuda.o -lcudart
*/



#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;


//kernel function to calculate the value of pi
__global__ void estimatePI(double *valueX, double *valueY, int *countNum, int gridSize, int N) {
    
    // shared array for shared memory
    __shared__ int shMem[512];
    
    int tid = threadIdx.x;
    int bid = blockIdx.x * blockDim.x;

    // the index and the jump from the index is
    int arrayId = tid + bid;
    int jump = blockDim.x * gridSize;
    
    int cnt = 0;
    for(int i = arrayId; i < N; i += jump) {
        double x = valueX[i];
        double y = valueY[i];
        
        double x1 = x * x;
        double y1 = y * y;
        
        if(x1 + y1 <= 1.0) {
            cnt += 1;
        }
    }
    
    shMem[tid] = cnt;
    
    __syncthreads();
    
    if(tid == 0) {
        int numOfHits = 0;    
        for(int j = 0; j < blockDim.x; j++) {      
            numOfHits += shMem[j];    
        }
        countNum[blockIdx.x] = numOfHits;  
    }
}

double randomNum(){
    double pt = (double)rand()/RAND_MAX;

    return pt;
}

int main() {

    int N = 10000000;
    //cout<<"Enter the value of N: ";
    //cin>>N;

    // allocate space to hold random values  
    double *arrayX = (double*) malloc(N * sizeof(double));
    double *arrayY = (double*) malloc(N * sizeof(double));
    
    //Initialize vector with random values    
    for (int i = 0; i < N; ++i) {
        arrayX[i] = randomNum();
        arrayY[i] = randomNum();
    }
    
    // Send random values to the GPU    
    double* valueX;
    double* valueY;
    
    // allocate the array values in the device
    hipMalloc(&valueX, N * sizeof(double));
    hipMalloc(&valueY, N * sizeof(double));
    
    // copy the array values in the device
    hipMemcpy(valueX, arrayX, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(valueY, arrayY, N * sizeof(double), hipMemcpyHostToDevice);
    
    // Launch kernel to count samples that fell inside unit circle    
    int numOfThreads = 512;
    int gridSize = N / (1000 * numOfThreads);
    int blockSize = gridSize * sizeof(int);
    
    int *countNum;
    
    hipMalloc(&countNum, blockSize);
    
    // call the kernel function
    estimatePI<<<gridSize, numOfThreads>>>(valueX, valueY, countNum, gridSize, N);
    
    int* res = new int[gridSize];
    // copy the array values back to the host
    hipMemcpy(res, countNum, blockSize, hipMemcpyDeviceToHost);
    
    int cnt = 0;
    for (int i = 0 ; i < gridSize; i++) {
        cnt += res[i];
    }
    
    hipFree(valueX);
    hipFree(valueY);
    hipFree(countNum);

    double pi_calculated = ((double)cnt / (double)N) * 4.0;
    printf("The value of PI by calculation is: %f \n", pi_calculated);


    return 0;
}
