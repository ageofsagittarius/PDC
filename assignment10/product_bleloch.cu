

#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

__global__ void prefixScanBleloch(int* input, int* output, int* pSum, int *sum, int n, int* up)
{
    int blockID = blockIdx.x * blockDim.x;
    int threadID = threadIdx.x;
    
    int arrayID = blockID + threadID;
    int offset = 1;
    
    sum[2 * threadID]   = input[2 * arrayID];
    sum[2 * threadID + 1] = input[2 * arrayID + 1];
    
    // Upsweep
    for(int i = n / 2; i > 0; i = i / 2){
        __syncthreads();
        if(threadID < i){
            int id1 = offset * (2 * threadID + 1) - 1;
            int id2 = offset * (2 * threadID + 2) - 1;
            sum[id2] *= sum[id1];
        }
        offset = offset * 2;
    }
    
    up[2 * arrayID] = sum[2 * threadID];
    up[2 * arrayID + 1] = sum[2 * threadID + 1];
    
    if(threadID == 0) {
        if(pSum != NULL){
            pSum[blockIdx.x] = sum[n-1];
        }
        sum[n-1] = 1;
    }
    
    // Downsweep
    for (int i = 1; i < n; i = i * 2){
        offset = offset / 2;
        __syncthreads();
        if (threadID < i){
            int id1 = offset * (2 * threadID + 1) - 1;
            int id2 = offset * (2 * threadID + 2) - 1;
            int temp = sum[id1];
            sum[id1]  = sum[id2];
            sum[id2] *= temp;
        }
    }
    __syncthreads();

    output[2 * arrayID] = sum[2 * threadID];
    output[2 * arrayID + 1] = sum[2 * threadID + 1];
}

int main()
{
    // the size of the array
    int size = 8;
    
    printf("The size of the array is: %d \n", size);
    
    //Create pointers and allocate arrays
    int *input = (int*) malloc(size * sizeof(int));
    int *output = (int*) malloc(size * sizeof(int));
    
    int *up = (int*) malloc(size * sizeof(int));
    
    for(int i = 0; i < size; i++){
        input[i] = i + 1;
    }
    
    printf("\nThe input array is: \n");
    for(int i = 0; i < size; i++){
        cout << input[i] << "\t";
    }
    cout<<endl;
    
    int numOfThreads = 1024;
    int blockSize = 2 * numOfThreads;
    int n;
    if(size % blockSize == 0){
        n = size;
    }
    else{
        n = (1 + size / blockSize) * blockSize;
    }
    int gridSize = n/blockSize;
    
    int *arr, *ans, *sum, *newSum, *upPtr;
    hipMalloc(&arr, sizeof(int) * size);
    hipMalloc(&upPtr, sizeof(int) * size);
    hipMalloc(&ans, sizeof(int) * n);
    hipMalloc(&sum, sizeof(int) * gridSize);
    hipMalloc(&newSum, sizeof(int) * gridSize);
    
    printf("\nNow Copying the memory to device.\n");
    hipMemcpy(arr, input, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(ans, output, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(upPtr, up, sizeof(int) * n, hipMemcpyHostToDevice);
    
    printf("\nNow performing scan using cuda\n");
    // perform scan using cuda
    prefixScanBleloch<<<gridSize, numOfThreads>>>(arr, ans, sum, newSum, blockSize, upPtr);
    hipDeviceSynchronize();
    
    printf("\nCopying the results back.\n");
    // copy scan result back to output
    hipMemcpy(output, ans, sizeof(int)*size, hipMemcpyDeviceToHost);
    
    hipMemcpy(up, upPtr, sizeof(int)*size, hipMemcpyDeviceToHost);
    
    
    printf("\nThe result of the upsweep is: \n");
    for(int i = 0; i < size; i++){
        cout<<up[i]<<" \t";
    }
    cout<<endl;
    
    printf("\nThe resultant prefixSum array is: \n");
    for(int i = 0; i < size; i++){
        cout<<output[i]<<" \t";
    }
    cout<<endl<<endl;

    //Free the allocated memory
    hipFree(arr);
    hipFree(ans);
    hipFree(sum);
    hipFree(newSum);
    
    free(input);
    free(output);
    
    return 0;
}
