

#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

__global__ void prefixScanHillis(int* input, int* output, int offset, int size){

    // calculating the index of the array
    int idx = (gridDim.x * blockDim.x) * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    
    // checking for the index being out of bounds
    if (idx < size){
        output[idx] = input[idx];
        if (idx >= offset){
            output[idx] *= input[idx - offset];
        }
    } 
}

int main(){

    // the size of the array
    int size = 8;
    
    printf("The size of the array is: %d \n", size);

    //Create pointers and allocate arrays
    int *input = (int*) malloc(size * sizeof(int));
    int *output = (int*) malloc(size * sizeof(int));
    
    int *arr, *ans;
    
    hipMalloc(&arr, size * sizeof(int));
    hipMalloc(&ans, size * sizeof(int));
    
    //Fill the input array with numbers
    for(int i = 0; i < size; i++){ 
        input[i] = i + 1;
    }
    
    printf("\nThe input array is: \n");
    for(int i = 0; i < size; i++){
        cout<<input[i]<<" \t";
    }
    cout<<endl;
    
    printf("\nNow Copying the memory to device.\n");
    hipMemcpy(arr, input, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(ans, output, sizeof(int) * size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    int numOfThreads = 1024;
    int blockSize = 2 * numOfThreads;
    int n;
    if(size % blockSize == 0){
        n = size;
    }
    else{
        n = (1 + size / blockSize) * blockSize;
    }
    int gridSize = n/blockSize;
    
    int offset = 1, limit = 2 * size;
    //int gridSize = 1, blockSize = size;
    int* temp;
    while(offset < limit){

        prefixScanHillis <<<gridSize, numOfThreads>>>(arr, ans, offset, size);
        hipDeviceSynchronize();

        offset = offset * 2;

        temp = arr;
        arr = ans;
        ans = temp;
    }

    printf("\nCopying the results back.\n");
    hipMemcpy(output, ans, size * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("\nThe resultant prefixSum array is: \n");
    for(int i = 0; i < size; i++){
        cout<<output[i]<<" \t";
    }
    cout<<endl<<endl;

    //Free the allocated memory
    hipFree(arr);
    hipFree(ans);
    
    free(input);
    free(output);
    
    return 0;
}
